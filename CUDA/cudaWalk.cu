// Ground Zero


#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <ctime>
#include <hiprand/hiprand_kernel.h> // **

using namespace std;

__global__
void loop(double* array,  long N, long R, double pi, double x, double t)
{

    int index = threadIdx.x;
    // int stride = blockDim.x;

    // int tId = index + (blockIdx.x * stride);
    hiprandState state;
    // "index" alone works for us - no need for "grid-stride" loops, as we are limiting to just 512 elements in the array
    // And, each element is handled by individual thread (ideally) -- Nope, not working as intended
    hiprand_init((unsigned long long)clock(), index, 1, &state); // ** Set offset = 1
    double dx = 2.0/R;
    
    int T = (int)(10*hiprand_uniform(&state))%4;

    // __syncthreads(); // Did not help
    while (x < (pi - (1.0/R))) {
        if (T == 0) {
            x += dx;
            if (x + dx > pi)
                x = 2*pi - x;
        }
        else if (T == 1)
            x = fabs(x - dx);
        t = t + 1;
        T = (int)(10*hiprand_uniform(&state))%4;
    }
    // __syncthreads(); // Did not help
    
    // for (int i = index; i < N; i += stride)
        array[index] = t*(1.0/8760);
}

int main()
{
    double pi = acos(-1);

    long rand_size = 1<<9; // 2^9 = 512 - Number of times, the walk shall be simulated, per "R" value - size of the "store" array
    double *store; // Array for storing the output, from each simulation, per "R" value
    hipMallocManaged(&store, rand_size*sizeof(double)); // Allocating memory in "Unified Storage" for "store"

    std::setprecision(20);
    
    // Input Radius
    double R = 0;
    cout << "\nR = ";
    cin >> R;
    cout << "\n";

    // Theoretical Value
    long double th = pow((pi*R - (1.0/R)), 2)/(2*8760);

    // Calling the kernel - 1 Block * 512 Threads = 512 Threads
    // Each thread shall simulate the Random Walk once, for a total of 512 outputs to the array
    loop<<<1, 512>>>(store, rand_size, R, pi, 0, 0);
    hipDeviceSynchronize(); // Waiting for GPU to finish

    // Summing the terms in the array is trivial and fast on the CPU
    double sum = 0;
    for (int i = 0 ; i < rand_size ; i++) {
        cout << store[i] << "\n";
        sum += store[i];
    }

    // Value, from the simulation
    double sim = sum/rand_size;

    // Output the values
    cout << "\nSimulated: " << sim << "\nTheoretical: " << th << "\nError Percentage: " << fabs(th - sim)/(0.01*th) << "\n";
    hipFree(store);

    return 0; // Have to check, if this is really important to keep
}
